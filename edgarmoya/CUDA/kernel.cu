#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#define TPB 64
#define ATOMIC 1 // 0 for non-atomic addition

__global__
void dotKernel(int *d_res, const int *d_a, const int *d_b, int n) {
    const int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= n) return;

    /*int prod = d_a[idx] * d_b[idx];
    if (ATOMIC) {
        atomicAdd(d_res, prod);
    } else {
        *d_res += prod;
    }*/

    const int s_idx = threadIdx.x;
    __shared__ int s_prod[TPB];
    s_prod[s_idx] = d_a[idx] * d_b[idx];
    __syncthreads();
    if (s_idx == 0) {
        int blockSum = 0;
        for (int j = 0; j < blockDim.x; ++j) {
            blockSum += s_prod[j];
        }

        if (ATOMIC) {
            atomicAdd(d_res, blockSum);
        } else {
            *d_res += blockSum;
        }
    }
}

void dotLauncher(int *res, const int *a, const int *b, int n) {
    int *d_res;
    int *d_a = 0;
    int *d_b = 0;

    hipMalloc(&d_res, sizeof(int));
    hipMalloc(&d_a, n*sizeof(int));
    hipMalloc(&d_b, n*sizeof(int));

    hipMemset(d_res, 0, sizeof(int));
    hipMemcpy(d_a, a, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n*sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start_gpu, end_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&end_gpu);

    // Capturar el tiempo de inicio en la GPU
    hipEventRecord(start_gpu);

    dotKernel<<<(n + TPB - 1)/TPB, TPB>>>(d_res, d_a, d_b, n);

    // Capturar el tiempo de finalización en la GPU
    hipEventRecord(end_gpu);
    hipEventSynchronize(end_gpu);

    // Calcular la duración e imprimir el resultado en milisegundos
    float duration_gpu;
    hipEventElapsedTime(&duration_gpu, start_gpu, end_gpu);
    std::cout << "Tiempo de ejecución en GPU: " << duration_gpu << " ms" << std::endl;

    hipMemcpy(res, d_res, sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(d_res);
    hipFree(d_a);
    hipFree(d_b);

    // Liberar eventos
    hipEventDestroy(start_gpu);
    hipEventDestroy(end_gpu);
}
