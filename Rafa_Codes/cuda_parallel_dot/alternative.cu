#include "hip/hip_runtime.h"
/*
    Alternativa del CUDA by example cap 5
*/
#include <bits/stdc++.h>
#define imin(a, b) (a < b ? a : b)

const int N = 10000 * 1024;
const int threadsPerBlock = 64;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(int *a, int *b, int *c)
{
    __shared__ int cache[threadsPerBlock];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    long long temp = 0;
    while (tid < N)
    {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    // set the cache values
    cache[cacheIndex] = temp;

    // synchronize threads in this block
    __syncthreads();

    // for reductions, threadsPerBlock must be a power of 2
    // because of the following code
    long long i = blockDim.x / 2;
    while (i != 0)
    {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }
    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}

long long main(void)
{
    
    long long *a, *b, c, *partial_c;
    long long *dev_a, *dev_b, *dev_partial_c;
    
    // allocate memory on the CPU side
    a = (long long *)malloc(N * sizeof(long long));
    b = (long long *)malloc(N * sizeof(long long));
    partial_c = (long long *)malloc(blocksPerGrid * sizeof(long long));
    
    // allocate the memory on the GPU
    hipMalloc((void **)&dev_a, N * sizeof(long long));
    hipMalloc((void **)&dev_b, N * sizeof(long long));
    hipMalloc((void **)&dev_partial_c, blocksPerGrid * sizeof(long long));
    
    // fill in the host memory with data
    for (long long i = 0; i < N; i++)
    {
        a[i] = 1;
        b[i] = 1;
    }
    
    // copy the arrays ‘a’ and ‘b’ to the GPU
    hipMemcpy(dev_a, a, N * sizeof(long long), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(long long), hipMemcpyHostToDevice);

    auto start = std::chrono::steady_clock::now();

    dot<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);

    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost);

    // finish up on the CPU side
    c = 0;
    for (int i = 0; i < blocksPerGrid; i++)
    {
        c += partial_c[i];
    }

    auto end = std::chrono::steady_clock::now();
    auto elapsedSequential = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    printf("Tiempo de ejecución paralelo: %ld ms\n", elapsedSequential.count());

    printf( "GPU value %d = %d?\n", c, 2 * sum_squares( (int)(N - 1) ) );


    auto start_s = std::chrono::steady_clock::now();
    int cpu_res = 0;
    for (int i = 0; i < N; ++i)
    {
        cpu_res += a[i] * b[i];
    }
    printf("cpu result = %d\n", cpu_res);
    auto end_s = std::chrono::steady_clock::now();
    auto elapsedSequential_s = std::chrono::duration_cast<std::chrono::milliseconds>(end_s - start_s);
    printf("Tiempo de ejecución secuencial: %ld ms\n", elapsedSequential_s.count());

    // free memory on the GPU side
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_partial_c);
    
    // free memory on the CPU side
    free(a);
    free(b);
    free(partial_c);
}